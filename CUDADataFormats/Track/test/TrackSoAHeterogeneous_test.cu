#include "hip/hip_runtime.h"
#include "CUDADataFormats/Track/interface/TrackSoAHeterogeneousT_test.h"
#include "HeterogeneousCore/CUDAUtilities/interface/OneToManyAssoc.h"

namespace testTrackSoAHeterogeneousT {

  __global__ void fill(pixelTrack::TrackSoAView tracks_view) {
    int i = threadIdx.x;
    if (i == 0) {
      tracks_view.nTracks() = 420;
    }

    for (int j = i; j < tracks_view.metadata().size(); j += blockDim.x) {
      tracks_view[j].pt() = (float)j;
      tracks_view[j].eta() = (float)j;
      tracks_view[j].chi2() = (float)j;
      tracks_view[j].quality() = (uint8_t)j % 256;
      tracks_view[j].nLayers() = j % 128;
      tracks_view.hitIndices().off[j] = j;
    }
  }

  __global__ void verify(pixelTrack::TrackSoAView tracks_view) {
    int i = threadIdx.x;

    if (i == 0) {
      printf("SoA size: % d, block dims: % d\n", tracks_view.metadata().size(), blockDim.x);
      assert(tracks_view.nTracks() == 420);
    }
    for (int j = i; j < tracks_view.metadata().size(); j += blockDim.x) {
      assert(abs(tracks_view[j].pt() - (float)j) < .0001);
      assert(abs(tracks_view[j].eta() - (float)j) < .0001);
      assert(abs(tracks_view[j].chi2() - (float)j) < .0001);
      assert(tracks_view[j].quality() == j % 256);
      assert(tracks_view[j].nLayers() == j % 128);
      assert(tracks_view.hitIndices().off[j] == j);
    }
  }

  void runKernels(pixelTrack::TrackSoAView tracks_view, uint32_t soaSize) {
    fill<<<1, 1024>>>(tracks_view);
    hipDeviceSynchronize();
    verify<<<1, 1024>>>(tracks_view);
  }

}  // namespace testTrackSoAHeterogeneousT
