#include "hip/hip_runtime.h"
#include "CUDADataFormats/Track/interface/TrackSoAHeterogeneousDevice.h"
#include "CUDADataFormats/Track/interface/TrackSoAHeterogeneousHost.h"
#include "HeterogeneousCore/CUDAUtilities/interface/OneToManyAssoc.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"

namespace testTrackSoAHeterogeneousT {

  __global__ void fill(pixelTrack::TrackSoAView tracks_view) {
    int i = threadIdx.x;
    if (i == 0) {
      tracks_view.nTracks() = 420;
    }

    for (int j = i; j < tracks_view.metadata().size(); j += blockDim.x) {
      tracks_view[j].pt() = (float)j;
      tracks_view[j].eta() = (float)j;
      tracks_view[j].chi2() = (float)j;
      tracks_view[j].quality() = (uint8_t)j % 256;
      tracks_view[j].nLayers() = j % 128;
      tracks_view.hitIndices().off[j] = j;
    }
  }

  // TODO: Use TrackSoAConstView when https://github.com/cms-sw/cmssw/pull/39919 is merged
  __global__ void verify(pixelTrack::TrackSoAView tracks_view) {
    int i = threadIdx.x;

    if (i == 0) {
      printf("SoA size: % d, block dims: % d\n", tracks_view.metadata().size(), blockDim.x);
      assert(tracks_view.nTracks() == 420);
    }
    for (int j = i; j < tracks_view.metadata().size(); j += blockDim.x) {
      assert(abs(tracks_view[j].pt() - (float)j) < .0001);
      assert(abs(tracks_view[j].eta() - (float)j) < .0001);
      assert(abs(tracks_view[j].chi2() - (float)j) < .0001);
      assert(tracks_view[j].quality() == j % 256);
      assert(tracks_view[j].nLayers() == j % 128);
      assert(tracks_view.hitIndices().off[j] == j);
    }
  }

  void runKernels(pixelTrack::TrackSoAView tracks_view, hipStream_t stream) {
    fill<<<1, 1024, 0, stream>>>(tracks_view);
    cudaCheck(hipGetLastError());
    cudaCheck(hipDeviceSynchronize());

    verify<<<1, 1024, 0, stream>>>(tracks_view);
    cudaCheck(hipGetLastError());
    cudaCheck(hipDeviceSynchronize());
  }

}  // namespace testTrackSoAHeterogeneousT
